#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/gate_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void GateForward(const int n, const Dtype* in,
    const Dtype* mask,
    Dtype* out, const int channels) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * mask[index % channels];
  }
}

template <typename Dtype>
void GateLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();


	const Dtype* mask = this->blobs_[0]->gpu_data();
	GateForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
		count, bottom_data, mask, top_data,bottom[0]->channels());
	CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
__global__ void GateBackward(const int n, const Dtype* in_diff,
    const Dtype* mask,  Dtype* out_diff, bool blob_diff, const int channels) {
  CUDA_KERNEL_LOOP(index, n) {
	if (!blob_diff){
		out_diff[index] = in_diff[index] * mask[index % channels];
	}
	else 
	{
		out_diff[index % channels] += in_diff[index] * mask[index];
	}
  }
}

template <typename Dtype>
void GateLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  if (this->param_propagate_down_[0]) {
      const Dtype* bottom_data = bottom[0]->gpu_data();
	  Dtype* blob_diff = this->blobs_[0]->mutable_gpu_diff();
	  caffe_gpu_set(this->blobs_[0]->count(), Dtype(0.), blob_diff);
      const int count = bottom[0]->count();
      // NOLINT_NEXT_LINE(whitespace/operators)

        GateBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
             count, top_diff, bottom_data, blob_diff, true,bottom[0]->channels());
       
      CUDA_POST_KERNEL_CHECK;
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      const Dtype* mask = this->blobs_[0]->gpu_data();
      const int count = bottom[0]->count();
      // NOLINT_NEXT_LINE(whitespace/operators)

        GateBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
             count, top_diff, mask,  bottom_diff, false,bottom[0]->channels());
       
      CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(GateLayer);

}  // namespace caffe
