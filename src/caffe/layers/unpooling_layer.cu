#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/unpooling_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void UnpoolingForward(const int channels_, const int height_, const int width_,
    const Dtype* bottom_data,const Dtype* mask_data, Dtype* top_data, int bdos, int mdos, int topos) {
    CUDA_KERNEL_LOOP(c, channels_) {
      for (int ph = 0; ph < height_; ++ph) {
        for (int pw = 0; pw < width_; ++pw) {
          const int index = ph * width_ + pw;
          top_data[static_cast<int>(mask_data[index+c*mdos]) + c * topos] = bottom_data[index+c*bdos];
        }
      }
    }
}


template <typename Dtype>
void UnPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* mask_data = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int top_count = top[0]->count();
  // Different pooling methods. We explicitly do the switch outside the for
  // loop to save time, although this results in more code.
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    // Initialize
UnpoolingForward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->num() * channels_), CAFFE_CUDA_NUM_THREADS>>>(
      bottom[0]->num() * channels_, height_, width_, bottom_data, mask_data, top_data, bottom[0]->offset(0, 1),bottom[1]->offset(0, 1),top[0]->offset(0, 1));
    CUDA_POST_KERNEL_CHECK;
    break;
  case PoolingParameter_PoolMethod_AVE:
    NOT_IMPLEMENTED;
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    NOT_IMPLEMENTED;
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
}

template <typename Dtype>
void UnPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    return;
}
INSTANTIATE_LAYER_GPU_FUNCS(UnPoolingLayer);

}  // namespace caffe
