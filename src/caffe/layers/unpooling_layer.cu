#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/unpooling_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void UnpoolingForward(const int channels_, const int height_, const int width_,
    const Dtype* bottom_data,const Dtype* mask_data, Dtype* top_data, int bdos, int mdos, int topos) {
    CUDA_KERNEL_LOOP(c, channels_) {
      for (int ph = 0; ph < height_; ++ph) {
        for (int pw = 0; pw < width_; ++pw) {
          const int index = ph * width_ + pw;
          top_data[static_cast<int>(mask_data[index+c*mdos]) + c * topos] = bottom_data[index+c*bdos];
        }
      }
    }
}


template <typename Dtype>
void UnPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* mask_data = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int top_count = top[0]->count();

  const int count = bottom[0]->count();
  caffe_gpu_set(top_count, Dtype(0.), top_data);

  // Different pooling methods. We explicitly do the switch outside the for
  // loop to save time, although this results in more code.
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    // Initialize
	UnpoolingForward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->num() * channels_), CAFFE_CUDA_NUM_THREADS>>>(
      bottom[0]->num() * channels_, height_, width_, bottom_data, mask_data, top_data, bottom[0]->offset(0, 1),bottom[1]->offset(0, 1),top[0]->offset(0, 1));

   
    break;
  case PoolingParameter_PoolMethod_AVE:
    NOT_IMPLEMENTED;
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    NOT_IMPLEMENTED;
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
	
  }
  CUDA_POST_KERNEL_CHECK;
}



template <typename Dtype>
__global__ void UnpoolingBackward(const int channels_, const int height_, const int width_,
    Dtype* bottom_diff,const Dtype* mask_data, const Dtype* top_diff, int bdos, int mdos, int topos) {
    CUDA_KERNEL_LOOP(c, channels_) {
      for (int ph = 0; ph < height_; ++ph) {
        for (int pw = 0; pw < width_; ++pw) {
          const int index = ph * width_ + pw;
		  bottom_diff[index+c*bdos] += top_diff[static_cast<int>(mask_data[index+c*mdos]) + c * topos];
        }
      }
    }
}




template <typename Dtype>
void UnPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

	  if (!propagate_down[0]) {
		return;
	  }
	  
	  

	  const Dtype* top_diff = top[0]->gpu_diff();
	  const Dtype* mask_data = bottom[1]->gpu_data();
	  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
	  const int top_count = top[0]->count();

	  const int count = bottom[0]->count();
	  caffe_gpu_set(count, Dtype(0.), bottom_diff);


	  switch (this->layer_param_.pooling_param().pool()) {
	  case PoolingParameter_PoolMethod_MAX:
		// NOLINT_NEXT_LINE(whitespace/operators)
		UnpoolingBackward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->num() * channels_), CAFFE_CUDA_NUM_THREADS>>>(
		  bottom[0]->num() * channels_, height_, width_, bottom_diff, mask_data, top_diff, bottom[0]->offset(0, 1),bottom[1]->offset(0, 1),top[0]->offset(0, 1));
		break;
	  case PoolingParameter_PoolMethod_AVE:
		// NOLINT_NEXT_LINE(whitespace/operators)    
		NOT_IMPLEMENTED;
		break;
	  case PoolingParameter_PoolMethod_STOCHASTIC:
		// NOLINT_NEXT_LINE(whitespace/operators)
		NOT_IMPLEMENTED;
		break;
	  default:
		LOG(FATAL) << "Unknown pooling method.";
	  }
	  CUDA_POST_KERNEL_CHECK;

}
INSTANTIATE_LAYER_GPU_FUNCS(UnPoolingLayer);

}  // namespace caffe
