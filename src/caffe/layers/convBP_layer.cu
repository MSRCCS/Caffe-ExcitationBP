#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/convBP_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void pos_kernel_convBP(const int n, const Dtype* a, Dtype* b) {
  CUDA_KERNEL_LOOP(index, n) {
    if (a[index] > 0)
      b[index] = a[index];
  }
}

template <typename Dtype>
__global__ void div_r_kernel_convBP(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    if (b[index] != 0)
      y[index] = a[index] / b[index];
    else
      y[index] = 0;
  }
}


template <typename Dtype>
void ConvolutionBPLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  vector<int> weight_shape = this->blobs_[0]->shape();
  vector<int> top_diff_shape = top[0]->shape(); 
  vector<int> bottom_diff_shape = bottom[0]->shape(); 
  vector<int> bottom_org_shape = bottom[1]->shape(); 





  bool bMatch = weight_shape[0]*weight_shape[2]*weight_shape[3]==top_diff_shape[1]*top_diff_shape[2]*top_diff_shape[3] 
       && weight_shape[0]*weight_shape[2]*weight_shape[3]==bottom_org_shape[1]*bottom_org_shape[2]*bottom_org_shape[3] // Match of original bottom
             && weight_shape[1]==bottom_diff_shape[1]; // Match of the diff to be propaged down

  // ToDo: Move the monitor in once code is stable
  if ( false )
  {
     LOG(INFO)<<"Name of layer: " << this->layer_param_.name();
     LOG(INFO)<<"Shape of W_data:" << this->blobs_[0]->shape_string();
     LOG(INFO)<<"Shape of top: "<<top[0]->shape_string();
     for ( int s=0; s<(int)bottom.size(); s++ )
     {
        LOG(INFO)<<"Shape of bottom " << s <<": "<<bottom[s]->shape_string();
     }
  };
  if ( !bMatch ) 
  {
     //LOG(FATAL)<<"Mismatched shape for the ConvBP layer";
     LOG(WARNING)<<"Mismatched shape for the ConvBP layer";
  }
  // get the new weight W+
  const Dtype* W_data = this->blobs_[0]->gpu_data();
  Blob<Dtype> W_plus(this->blobs_[0]->shape());
  Dtype* W_plus_data = W_plus.mutable_gpu_data();
  caffe_gpu_set<Dtype>(W_plus.count(), Dtype(0), W_plus_data);
  pos_kernel_convBP<Dtype><<<CAFFE_GET_BLOCKS(W_plus.count()), CAFFE_CUDA_NUM_THREADS>>>(
        W_plus.count(), W_data, W_plus_data);

  LOG(INFO)<<"Copy W_data to W_plus";
  Blob<Dtype> NN(bottom[0]->shape());
  Dtype* NN_data = NN.mutable_gpu_data();
  
  //LOG(INFO)<<"This Num is: "<< this->num_<<" Group is: "<< this-> group_;
  //LOG(INFO)<<"Conv out channels is: "<< this->conv_out_channels_;
  //LOG(INFO)<<"Conv out spatial dimension is: "<< this->conv_out_spatial_dim_; 
  //LOG(INFO)<<"Kernel dimension is: "<< this->kernel_dim_; 


  // The following name follows the convention of the mirrored forward path 
  // on the classification path.
  // compute the normalization factor by forwardpassing using W+
  const Dtype* bottom_data = bottom[1]->gpu_data();
  for (int n = 0; n < this->num_; ++n) {
    this->forward_gpu_gemm(bottom_data + n * this->top_dim_, W_plus_data,
      NN_data + n * this->bottom_dim_);
  }




  //this->print_vector(bottom_data, bottom[1]->count() );
              //this->print_vector( W_plus_data, this->blobs_[0]->count() );
              //this->print_vector( NN_data, bottom[0]->count() );
  // do normalization

  const Dtype* top_diff = bottom[0]->gpu_data();
  div_r_kernel_convBP<Dtype><<<CAFFE_GET_BLOCKS(NN.count()), CAFFE_CUDA_NUM_THREADS>>>(
        NN.count(), top_diff, NN_data, NN_data);



              //LOG(INFO)<<"Done Compute normalization";
              //this->print_vector( NN_data, bottom[0]->count() );
  // do backward pass
  Dtype* bottom_diff = top[0]->mutable_gpu_data();
  for (int n = 0; n < this->num_; ++n) {
    this->backward_gpu_gemm(NN_data + n * this->bottom_dim_, W_plus_data,
      bottom_diff + n * this->top_dim_);
  }

              //LOG(INFO)<<"Done backward pass";
              //this->print_vector( bottom_diff, top[0]->count() );

  // multiply the bottom data
  caffe_gpu_mul<Dtype>(top[0]->count(), bottom_diff, bottom_data, bottom_diff);

              //LOG(INFO)<<"Multiply bottom data";
              //this->print_vector( bottom_diff, top[0]->count() );


















}

template <typename Dtype>
void ConvolutionBPLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
}

template <typename Dtype>
__global__ void pos_kernel(const int n, const Dtype* a, Dtype* b) {
  CUDA_KERNEL_LOOP(index, n) {
    if (a[index] > 0)
      b[index] = a[index];
  }
}

template <typename Dtype>
__global__ void div_r_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    if (b[index] != 0)
      y[index] = a[index] / b[index];
    else
      y[index] = 0;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionBPLayer);

}  // namespace caffe
